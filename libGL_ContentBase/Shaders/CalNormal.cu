#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void ComputeNormalMap_kernel(int width, int height, float3 *vertexMap, float3 *normalMap)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = y * width +x;
    if( x < 1 || x >= width - 1 || y < 1 || y >= height - 1 ){
        normalMap[idx] = make_float3(0.f,0.f,0.f);
        return;
    }
    
    float3 left =  vertexMap[idx - 1];
    float3 right = vertexMap[idx + 1];
    float3 upper = vertexMap[idx - width];
    float3 lower = vertexMap[idx + width];
    
    float3 normal;
    if(left.z == 0 || right.z == 0 || upper.z == 0 || lower.z == 0 ){
        normal.x = normal.y = normal.z = 0;
    }else{
    
        float3 hor = left - right;
        float3 ver = upper - lower;
        
        normal = cross(hor, ver);
        normal = normalize( normal );
        
        if (normal.z > 0){
            normal *= -1;
        }
        
    }
    
    normalMap[idx] = normal;
}

void ComputeNormalMap(int width, int height, float3 *vertexMap, float3 *normalMap)
{
    dim3 threads(16, 16);
    dim3 blocks( (width + threads.x-1) / threads.x, (height + threads.y-1) / threads.y );
    ComputeNormalMap_kernel<<<blocks, threads>>>( width, height, vertexMap, normalMap );
    hipDeviceSynchronize();
}
